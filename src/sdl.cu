#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : sdl.cu
 Author      : omikronsc
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <vector>
#include <algorithm>
#include <stdlib.h>
#include <SDL2/SDL.h>
#include <SDL2/SDL2_gfxPrimitives.h>
#include "triangulation/s_hull_pro.h"

using namespace std;

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
	unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < vectorSize)
		data[idx] = 1.0 / data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size) {
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void ** )&gpuData, sizeof(float) * size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float) * size, hipMemcpyHostToDevice));

	static const int BLOCK_SIZE = 256;
	const int blockCount = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>>(gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float) * size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

float *cpuReciprocal(float *data, unsigned size) {
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt)
		rc[cnt] = 1.0 / data[cnt];
	return rc;
}

void initialize(float *data, unsigned size) {
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5 * (i + 1);
}

int main(void) {
	if (SDL_Init(SDL_INIT_VIDEO) != 0) {
		std::cout << "SDL_Init Error: " << SDL_GetError() << std::endl;
		return 1;
	}
	SDL_Window *win = SDL_CreateWindow("Hello World!", 0, 0, 1920, 1080, SDL_WINDOW_FULLSCREEN);
	if (win == nullptr) {
		std::cout << "SDL_CreateWindow Error: " << SDL_GetError() << std::endl;
		SDL_Quit();
		return 1;
	}
	SDL_Renderer *ren = SDL_CreateRenderer(win, -1,
			SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
	if (ren == nullptr) {
		SDL_DestroyWindow(win);
		std::cout << "SDL_CreateRenderer Error: " << SDL_GetError() << std::endl;
		SDL_Quit();
		return 1;
	}
	std::string imagePath = "/home/sebastian/data/genetmal/emily_bett_rickards.bmp";
//	std::string imagePath = "/home/sebastian/data/genetmal/summer_glau_original.bmp";
	SDL_Surface *bmp = SDL_LoadBMP(imagePath.c_str());
	if (bmp == nullptr) {
		SDL_DestroyRenderer(ren);
		SDL_DestroyWindow(win);
		std::cout << "SDL_LoadBMP Error: " << SDL_GetError() << std::endl;
		SDL_Quit();
		return 1;
	}
//	SDL_Texture *tex = SDL_CreateTextureFromSurface(ren, bmp);
//	SDL_FreeSurface(bmp);
//	if (tex == nullptr) {
//		SDL_DestroyRenderer(ren);
//		SDL_DestroyWindow(win);
//		std::cout << "SDL_CreateTextureFromSurface Error: " << SDL_GetError() << std::endl;
//		SDL_Quit();
//		return 1;
//	}

	int screenWidth, screenHeight;
	SDL_GetWindowSize(win, &screenWidth, &screenHeight);
	cout << "Window size is: " << screenWidth << "x" << screenHeight << endl;
	int imageWidth = bmp->w;
	int imageHeight = bmp->h;
	cout << "Image size is: " << imageWidth << "x" << imageHeight << endl;

	int width = min(screenWidth, imageWidth);
	int height = min(screenHeight, imageHeight);

	cout << "pixelFormat is " << SDL_GetPixelFormatName(bmp->format->format) << endl;

	Uint8 *pixels = new Uint8[width * height * 4];

	SDL_Texture* texture = SDL_CreateTexture(ren, SDL_PIXELFORMAT_ARGB8888,
			SDL_TEXTUREACCESS_STREAMING, width, height);

	struct Point {
		int x, y, r, g, b;
	};

	vector<Point> points;
	const int POINTS_NUM = 1000;
	const int MARGIN = 2;
	Point p;
	for (int i = 0; i < POINTS_NUM; i++) {
		p.x = rand() % (width - 2 * MARGIN) + MARGIN;
		p.y = rand() % (height - 2 * MARGIN) + MARGIN;
		p.r = rand() % 256;
		p.g = rand() % 256;
		p.b = rand() % 256;
		points.push_back(p);
	}

	std::vector<Shx> pts, hull;
	Shx pt;
	for (int v = 0; v < POINTS_NUM; v++) {
		pt.id = v;
		pt.r = points[v].x;
		pt.c = points[v].y;

		pts.push_back(pt);
	}
	sort(pts.begin(), pts.end(), pointSortPredicate);
	vector<Shx>::iterator newEnd = unique(pts.begin(), pts.end(), pointComparisonPredicate);
	pts.resize(newEnd - pts.begin());
	vector<Triad> triads;
	s_hull_pro(pts, triads);

	SDL_LockSurface(bmp);

	SDL_Event e;
	bool quit = false;
	while (!quit) {
		while (SDL_PollEvent(&e)) {
			if (e.type == SDL_QUIT) {
				quit = true;
			}
			if (e.type == SDL_KEYDOWN && e.key.keysym.sym == SDLK_ESCAPE) {
				quit = true;
			}
		}

		Uint64 begin = SDL_GetPerformanceCounter();

		SDL_RenderClear(ren);
		for (int x = 0; x < width; x++) {
			for (int y = 0; y < height; y++) {
				Uint32 * targetPixel;
				int pixelPosition = y * bmp->pitch + x * bmp->format->BytesPerPixel;
				targetPixel = (Uint32*) ((Uint8 *) bmp->pixels + pixelPosition);
				Uint8 r, g, b;
				SDL_GetRGB(*targetPixel, bmp->format, &r, &g, &b);

				const unsigned int offset = (y * imageWidth + x) * 4;
				pixels[offset] = b;
				pixels[offset + 1] = g;
				pixels[offset + 2] = r;
				pixels[offset + 3] = 0xFF;

//				SDL_SetRenderDrawColor(ren, r, g, b, a);
//				SDL_RenderDrawPoint(ren, x, y);
			}
		}

		SDL_UpdateTexture(texture, NULL, pixels, width * 4);
		SDL_RenderCopy(ren, texture, NULL, NULL);

		for (auto triad : triads) {
			int x1 = points[triad.a].x;
			int y1 = points[triad.a].y;
			int x2 = points[triad.b].x;
			int y2 = points[triad.b].y;
			int x3 = points[triad.c].x;
			int y3 = points[triad.c].y;

			trigonRGBA(ren, x1, y1, x2, y2, x3, y3, 0, 0, 0, 0xFF);
		}

		for (int i = 0; i < POINTS_NUM; i++) {
			Point *p = &points[i];
			boxRGBA(ren, p->x - MARGIN, p->y - MARGIN, p->x + MARGIN, p->y + MARGIN, p->r, p->g,
					p->b, 0xFF);
		}

		//Update the screen
		SDL_RenderPresent(ren);

		Uint64 t = (SDL_GetPerformanceCounter() - begin) * 1000 / SDL_GetPerformanceFrequency();
		cout << t << "ms" << endl;
	}

	SDL_UnlockSurface(bmp);

	delete pixels;

	SDL_DestroyRenderer(ren);
	SDL_DestroyWindow(win);
	SDL_Quit();
}

int main2(void) {
	static const int WORK_SIZE = 65530;
	float *data = new float[WORK_SIZE];

	initialize(data, WORK_SIZE);

	float *recCpu = cpuReciprocal(data, WORK_SIZE);
	float *recGpu = gpuReciprocal(data, WORK_SIZE);
	float cpuSum = std::accumulate(recCpu, recCpu + WORK_SIZE, 0.0);
	float gpuSum = std::accumulate(recGpu, recGpu + WORK_SIZE, 0.0);

	/* Verify the results */
	std::cout << "gpuSum = " << gpuSum << " cpuSum = " << cpuSum << std::endl;

	/* Free memory */
	delete[] data;
	delete[] recCpu;
	delete[] recGpu;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement,
		hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at "
			<< file << ":" << line << std::endl;
	exit(1);
}

